#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES 
#define KERNEL_SIZE 3
#include "canny.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

void apply_canny(uint8_t* final_pixels, const uint8_t* ori_pixels, int weak_threshold, int strong_threshold, int image_width, int image_height, int thd_per_blk) {

	// gaussian kernel
	const double gaussian_kernel[9] = {
		1,2,1,
		2,4,2,
		1,2,1
	};
	const int8_t sobel_kernel_x[] = {   -1, 0, 1,
										-2, 0, 2,
										-1, 0, 1 };
	const int8_t sobel_kernel_y[] = {    1, 2, 1,
										 0, 0, 0,
										-1,-2,-1 };
	/* kernel execution configuration parameters */
	const int num_blks = (image_height * image_width) / thd_per_blk;
	const int grid = 0;

	/* device buffers */
	uint8_t* in, * out;
	double* gradient_pixels;
	double* max_pixels;
	uint8_t* segment_pixels;
	double* gaussian_kernel_gpu;
	int8_t* sobel_kernel_x_gpu;
	int8_t* sobel_kernel_y_gpu;
	uint8_t* final_result;

	float elapsed = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);  //start timer

	/* allocate device memory */
	hipMalloc((void**)&in, sizeof(uint8_t) * image_height * image_width);
	hipMalloc((void**)&out, sizeof(uint8_t) * image_height * image_width);
	hipMalloc((void**)&gradient_pixels, sizeof(double) * image_height * image_width);
	hipMalloc((void**)&final_result, sizeof(uint8_t) * image_height * image_width);
	hipMalloc((void**)&max_pixels, sizeof(double) * image_height * image_width);
	hipMalloc((void**)&segment_pixels, sizeof(uint8_t) * image_height * image_width);
	hipMalloc((void**)&gaussian_kernel_gpu, sizeof(double) * KERNEL_SIZE * KERNEL_SIZE);
	hipMalloc((void**)&sobel_kernel_x_gpu, sizeof(int8_t) * 3 * 3);
	hipMalloc((void**)&sobel_kernel_y_gpu, sizeof(int8_t) * 3 * 3);

	/* data transfer image pixels to device */
	hipMemcpy(in, ori_pixels, image_height * image_width * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(gaussian_kernel_gpu, gaussian_kernel, sizeof(double) * KERNEL_SIZE * KERNEL_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(sobel_kernel_x_gpu, sobel_kernel_x, sizeof(int8_t) * KERNEL_SIZE * KERNEL_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(sobel_kernel_y_gpu, sobel_kernel_y, sizeof(int8_t) * KERNEL_SIZE * KERNEL_SIZE, hipMemcpyHostToDevice);

	/* run canny edge detection core - CUDA kernels */
	/* use streams to ensure the kernels are in the same task */
	hipStream_t stream;
	hipStreamCreate(&stream);

	// 1. gaussian filter
	apply_gaussian_filter << <num_blks, thd_per_blk, grid, stream >> > (out, in, image_width, image_height, gaussian_kernel_gpu);
	// 2.apply sobel kernels
	apply_sobel_filter << <num_blks, thd_per_blk, grid, stream >> > (gradient_pixels, segment_pixels, out,
		image_width, image_height, sobel_kernel_x_gpu, sobel_kernel_y_gpu);
	hipMemcpy(max_pixels, gradient_pixels, image_height * image_width * sizeof(double), hipMemcpyDeviceToDevice);
	// 3. local maxima: non maxima suppression
	apply_non_max_suppression << <num_blks, thd_per_blk, grid, stream >> > (max_pixels, gradient_pixels, segment_pixels, image_width, image_height);
	// 4. double threshold
	apply_double_threshold << <num_blks, thd_per_blk, grid, stream >> > (out,max_pixels,strong_threshold,weak_threshold, image_width, image_height);
	// 5. edges with hysteresis
	hipMemcpy(final_result, out, image_height * image_width * sizeof(uint8_t), hipMemcpyDeviceToDevice);
	apply_edge_hysteresis << <num_blks, thd_per_blk, grid, stream >> > (final_result, out, image_width, image_height);

	/* wait for everything to finish */
	hipDeviceSynchronize();

	/* copy result back to the host */
	hipMemcpy(final_pixels, final_result, image_width * image_height * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(in);
	hipFree(out);
	hipFree(gradient_pixels);
	hipFree(max_pixels);
	hipFree(segment_pixels);
	hipFree(gaussian_kernel_gpu);
	hipFree(sobel_kernel_x_gpu);
	hipFree(sobel_kernel_y_gpu);
	hipFree(final_result);

	hipEventRecord(stop, 0); //end timer
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("The elapsed time in gpu was %.2f ms\n", elapsed);
}

__global__ void apply_gaussian_filter(uint8_t* out_pixels, const uint8_t* in_pixels, int image_width, int image_height, double* gaussian_kernel)
{
	//determine id of thread which corresponds to an individual pixel
	int pixNum = blockIdx.x * blockDim.x + threadIdx.x;
	const int offset_xy = ((KERNEL_SIZE - 1) / 2);
	if (!(pixNum >= 0 && pixNum < image_height * image_width))
		return;

	//Apply Kernel to image
	double kernelSum = 0;
	double pixelVal = 0;
	for (int i = 0; i < KERNEL_SIZE; ++i) {
		for (int j = 0; j < KERNEL_SIZE; ++j) {
			//check edge cases, if within bounds, apply filter
			if (((pixNum + ((i - offset_xy) * image_width) + j - offset_xy) >= 0)
				&& ((pixNum + ((i - offset_xy) * image_width) + j - offset_xy) <= image_height * image_width - 1)
				&& (((pixNum % image_width) + j - offset_xy) >= 0)
				&& (((pixNum % image_width) + j - offset_xy) <= (image_width - 1))) {

				pixelVal += gaussian_kernel[i * KERNEL_SIZE + j] * in_pixels[pixNum + ((i - offset_xy) * image_width) + j - offset_xy];
				kernelSum += gaussian_kernel[i * KERNEL_SIZE + j];
			}
		}
	}
	out_pixels[pixNum] = (uint8_t)(pixelVal / kernelSum);
	
}
__global__ void apply_sobel_filter(double* gradient_pixels, uint8_t* segment_pixels, const uint8_t* in_pixels, int image_width, int image_height, int8_t* sobel_kernel_x, int8_t* sobel_kernel_y ) {
	//Sobel
	int pixNum = blockIdx.x * blockDim.x + threadIdx.x;
	if (!(pixNum >= 0 && pixNum < image_height * image_width))
		return;
	int x = pixNum % image_width;
	int y = pixNum / image_width;
	int offset_xy = 1;  // 3x3
	if (x < offset_xy || x >= image_width - offset_xy || y < offset_xy || y >= image_height - offset_xy)
		return;
	double convolve_X = 0.0;
	double convolve_Y = 0.0;
	int k = 0;
	int src_pos = x + (y * image_width);

	for (int ky = -offset_xy; ky <= offset_xy; ky++) {
		for (int kx = -offset_xy; kx <= offset_xy; kx++) {
			convolve_X += in_pixels[src_pos + (kx + (ky * image_width))] * sobel_kernel_x[k];
			convolve_Y += in_pixels[src_pos + (kx + (ky * image_width))] * sobel_kernel_y[k];
			k++;
		}
	}

	// gradient hypot & direction
	int segment = 0;

	if (convolve_X == 0.0 || convolve_Y == 0.0) {
		gradient_pixels[src_pos] = 0;
	}
	else {
		gradient_pixels[src_pos] = ((std::sqrt((convolve_X * convolve_X) + (convolve_Y * convolve_Y))));
		double theta = std::atan2(convolve_Y, convolve_X);  // radians. atan2 range: -PI,+PI,   // theta : 0 - 2PI
		theta = theta * (360.0 / (2.0 * M_PI));  // degrees

		if ((theta <= 22.5 && theta >= -22.5) || (theta <= -157.5) || (theta >= 157.5))
			segment = 1;  // "-"
		else if ((theta > 22.5 && theta <= 67.5) || (theta > -157.5 && theta <= -112.5))
			segment = 2;  // "/" 
		else if ((theta > 67.5 && theta <= 112.5) || (theta >= -112.5 && theta < -67.5))
			segment = 3;  // "|"
		else if ((theta >= -67.5 && theta < -22.5) || (theta > 112.5 && theta < 157.5))
			segment = 4;  // "\"  
	}
	segment_pixels[src_pos] = (uint8_t)segment;
		
}
__global__ void apply_non_max_suppression(double* max_pixels, double* gradient_pixels, uint8_t* segment_pixels, int image_width, int image_height) {
	int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (!(pos >= 0 && pos < image_height * image_width))
		return;
	switch (segment_pixels[pos]) {
	case 1:
		if (segment_pixels[pos - 1] >= gradient_pixels[pos] || gradient_pixels[pos + 1] > gradient_pixels[pos])
			max_pixels[pos] = 0;
		break;
	case 2:
		if (gradient_pixels[pos - (image_width - 1)] >= gradient_pixels[pos] || gradient_pixels[pos + (image_width - 1)] > gradient_pixels[pos])
			max_pixels[pos] = 0;
		break;
	case 3:
		if (gradient_pixels[pos - (image_width)] >= gradient_pixels[pos] || gradient_pixels[pos + (image_width)] > gradient_pixels[pos])
			max_pixels[pos] = 0;
		break;
	case 4:
		if (gradient_pixels[pos - (image_width + 1)] >= gradient_pixels[pos] || gradient_pixels[pos + (image_width + 1)] > gradient_pixels[pos])
			max_pixels[pos] = 0;
		break;
	default:
		max_pixels[pos] = 0;
		break;
	}

}
__global__ void apply_double_threshold(uint8_t* out, double* max_pixels, int strong_threshold, int weak_threshold, int image_width, int image_height) {
	int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (!(pos >= 0 && pos < image_height * image_width))
		return;
	if (max_pixels[pos] > strong_threshold)
		out[pos] = 255;      //absolutely edge
	else if (max_pixels[pos] > weak_threshold)
		out[pos] = 100;      //potential edge
	else
		out[pos] = 0;       //absolutely not edge
}
__global__ void apply_edge_hysteresis(uint8_t* out, uint8_t* in, int image_width, int image_height) {
	int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (!(pos >= 0 && pos < image_height * image_width))
		return;
	if (in[pos] == 100) {
		if (in[pos - 1] == 255 || in[pos + 1] == 255 ||
			in[pos - image_width] == 255 || in[pos + image_width] == 255 ||
			in[pos - image_width - 1] == 255 || in[pos - image_width + 1] == 255 ||
			in[pos + image_width - 1] == 255 || in[pos + image_width + 1] == 255)
			out[pos] = 255;
		else
			out[pos] = 0;
	}

}

